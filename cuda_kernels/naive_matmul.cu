/*
 * Naive CUDA matrix multiplication kernel
 * Each thread computes one element of the output matrix
 * This is the simplest implementation - not optimized for memory access patterns
 */

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <chrono>
using namespace std;

// CUDA error checking macro
#define CHECK_CUDA(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(error) << endl; \
            exit(1); \
        } \
    } while(0)

/**
 * Naive CUDA kernel for matrix multiplication
 * C = A * B where A is MxK, B is KxN, C is MxN
 * 
 * Each thread computes one element C[row][col]
 * Memory access pattern is not optimized - many global memory accesses
 */
__global__ void naive_matmul_kernel(
    const float* A, 
    const float* B, 
    float* C, 
    int M, int N, int K
) {
    // Calculate thread's position in the output matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds
    if (row < M && col < N) {
        float sum = 0.0f;
        
        // Compute dot product of row from A and column from B
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        
        C[row * N + col] = sum;
    }
}

/**
 * Host function to launch naive matrix multiplication
 */
void naive_matmul(
    const float* h_A, 
    const float* h_B, 
    float* h_C, 
    int M, int N, int K
) {
    // Device memory pointers
    float *d_A, *d_B, *d_C;
    
    // Calculate memory sizes
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    
    // Allocate device memory
    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
    
    // Launch configuration
    const int BLOCK_SIZE = 16;  // 16x16 thread blocks
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(
        (N + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (M + BLOCK_SIZE - 1) / BLOCK_SIZE
    );
    
    // Launch kernel
    naive_matmul_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Copy result back to host
    CHECK_CUDA(hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost));
    
    // Free device memory
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
}

/**
 * Benchmark function for naive implementation
 */
float benchmark_naive_matmul(int M, int N, int K, int num_runs = 10) {
    // Allocate host memory
    float* h_A = new float[M * K];
    float* h_B = new float[K * N];
    float* h_C = new float[M * N];
    
    // Initialize matrices with random values
    for (int i = 0; i < M * K; i++) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < K * N; i++) h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    
    // Warm up
    naive_matmul(h_A, h_B, h_C, M, N, K);
    
    // Benchmark
    auto start = chrono::high_resolution_clock::now();
    
    for (int run = 0; run < num_runs; run++) {
        naive_matmul(h_A, h_B, h_C, M, N, K);
    }
    
    auto end = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
    
    float avg_time_ms = duration.count() / (1000.0f * num_runs);
    
    // Calculate GFLOPS
    long long flops = 2LL * M * N * K;
    float gflops = flops / (avg_time_ms * 1e6);
    
    cout << "Naive CUDA - Size: " << M << "x" << N << "x" << K 
              << ", Time: " << avg_time_ms << " ms"
              << ", Performance: " << gflops << " GFLOPS" << endl;
    
    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    
    return avg_time_ms;
}

/**
 * Main function for testing
 */
int main() {
    cout << "=== Naive CUDA Matrix Multiplication Benchmark ===" << endl;
    
    // Get device properties
    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDeviceProperties(&prop, 0));
    cout << "GPU: " << prop.name << endl;
    cout << "Compute capability: " << prop.major << "." << prop.minor << endl;
    cout << "Global memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << endl;
    cout << "Shared memory per block: " << prop.sharedMemPerBlock / 1024 << " KB" << endl;
    cout << endl;
    
    // Test different matrix sizes
    int sizes[] = {64, 128, 256, 512, 1024};
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);
    
    for (int i = 0; i < num_sizes; i++) {
        int size = sizes[i];
        benchmark_naive_matmul(size, size, size);
    }
    
    return 0;
}
